#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include ""
#include <hip/hip_runtime.h>
#define pi 3.141592653589793
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
        }                                                                                          \
    } while (0)
using namespace std;

__device__ void full_model(double x, double y, double z, double u, double R, double* C, double* S, double* fac, double* f1, double* f2, double* f3, double* J1, double* J2, double* J3, double* J4, double* J5, double* J6) {
    double r = pow(x * x + y * y + z * z, 0.5);
    double miu = z / r;
    double lambda = 2 * pi * (y < 0) + pow(-1, (y < 0)) * acos(x / pow(x * x + y * y, 0.5));
    double P[41][41] = { 0 };//{1,0,miu,pow(1-miu*miu,0.5)};
    double nor, U_, dUdr_, dUdmiu_, dUdlambda_, d2Udr2_, d2Udmiu2_, d2Udlambda2_;
    P[0][0] = 1; P[1][0] = miu; P[1][1] = pow(1 - miu * miu, 0.5);
    double dP[40][41] = { 0 }, d2P[39][41] = { 0 };
    double dUdr = 0, dUdmiu = 0, dUdlambda = 0, U = 0, d2Udr2 = 0, d2Udmiu2 = 0, d2Udlambda2 = 0;
    for (int n = 2; n <= 40; n++) {
        for (int m = 0; m <= n; m++) {
            if (m == 0) {
                P[n][m] = ((2 * n - 1) * miu * P[n - 1][0] - (n - 1) * P[n - 2][0]) / n;
            }
            if (m == n) {
                P[n][m] = (2 * n - 1) * pow(1 - miu * miu, 0.5) * P[n - 1][n - 1];
            }
            if (m == (n - 1)) {
                P[n][m] = (2 * n - 1) * miu * P[n - 1][n - 1];
            }
            if (m != 0 && m != n && m != (n - 1)) {
                P[n][m] = (2 * n - 1) * miu * P[n - 1][m] / (n - m) - (n + m - 1) * P[n - 2][m] / (n - m);
            }
            if (n > 2) {
                dP[n - 1][m] = ((m - n) * P[n][m] + n * miu * P[n - 1][m]) / (1 - miu * miu);
            }
            if (n > 3) {
                d2P[n - 2][m] = ((-1 + n + 2 * miu * miu + 3 * (n - 2) * miu * miu + (n - 2) * (n - 2) * miu * miu) * P[n - 2][m] + (1 + m - n) * ((5 + 2 * (n - 2)) * miu * P[n - 1][m] + (m - n) * P[n][m])) / pow(miu * miu - 1, 2);
                if (m <= n - 2) {
                    nor = pow(((m == 0) * 1 + (m != 0) * 2) * (2 * n - 3) * fac[n - m - 2] / fac[n + m - 2], 0.5);
                    U_ = pow(R / r, n - 2) * nor * P[n - 2][m] * (C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) + S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda));
                    U = U + U_;

                    dUdr_ = pow(R / r, n - 2) * (n - 1) * nor * P[n - 2][m] * (C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) + S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda));
                    dUdmiu_ = pow(R / r, n - 2) * nor * dP[n - 2][m] * (C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) + S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda));
                    dUdlambda_ = pow(R / r, n - 2) * nor * P[n - 2][m] * (-C[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda) * m + S[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) * m);

                    dUdr = dUdr + dUdr_; dUdmiu = dUdmiu + dUdmiu_; dUdlambda = dUdlambda + dUdlambda_;
                    
                    d2Udr2_ = pow(R / r, n - 2) * (n - 1) * n * nor * P[n - 2][m] * (C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) + S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda));
                    d2Udmiu2_ = pow(R / r, n - 2) * nor * d2P[n - 2][m] * (C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) + S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda));
                    d2Udlambda2_ = pow(R / r, n - 2) * nor * P[n - 2][m] * (-C[(n + 3) * (n - 2) / 2 + m] * cos(m * lambda) * m * m - S[(n + 3) * (n - 2) / 2 + m] * sin(m * lambda) * m * m);

                    d2Udr2 = d2Udr2 + d2Udr2_; d2Udmiu2 = d2Udmiu2 + d2Udmiu2_; d2Udlambda2 = d2Udlambda2 + d2Udlambda2_;
                    

                }
            }
        }
    }
    U = (u / r) * (1 + U);

    dUdr = (-u / pow(r, 2)) * (dUdr + 1); dUdmiu = (u / r) * (dUdmiu); dUdlambda = (u / r) * dUdlambda;
    *f1 = dUdr * (x / r) + dUdmiu * (-x * z / pow(r, 3)) + dUdlambda * (-y / (x * x + y * y));
    *f2 = dUdr * (y / r) + dUdmiu * (-y * z / pow(r, 3)) + dUdlambda * (x / (x * x + y * y));
    *f3 = dUdr * (z / r) + dUdmiu * ((x * x + y * y) / pow(r, 3)) + 0;
    
    d2Udr2 = (u / pow(r, 3)) * (2 + d2Udr2); d2Udmiu2 = (u / r) * d2Udmiu2;
    d2Udlambda2 = (u / r) * d2Udlambda2;

    *J1 = d2Udr2 * (pow(x, 2) / pow(r, 2)) + dUdr * (-pow(x, 2) / pow(r, 3) + 1 / r) + d2Udmiu2 * pow(-x * z / pow(r, 3), 2) + dUdmiu * (3 * pow(x, 2) * z / pow(r, 5) - z / pow(r, 3)) + d2Udlambda2 * (pow(-y / (pow(x, 2) + pow(y, 2)), 2)) + dUdlambda * (2 * x * y / pow(pow(x, 2) + pow(y, 2), 2));
    *J2 = d2Udr2 * (x * y / pow(r, 2)) + dUdr * (-x * y / pow(r, 3)) + d2Udmiu2 * (-x * z / pow(r, 3) * (-y * z / pow(r, 3))) + dUdmiu * (3 * x * y * z / pow(r, 5)) + d2Udlambda2 * (-y / (pow(x, 2) + pow(y, 2)) * x / (pow(x, 2) + pow(y, 2))) + dUdlambda * ((pow(y, 2) - pow(x, 2)) / pow(pow(x, 2) + pow(y, 2), 2));
    *J3 = d2Udr2 * (x * z / pow(r, 2)) + dUdr * (-x * z / pow(r, 3)) + d2Udmiu2 * (-x * z / pow(r, 3) * (-pow(z, 2) / pow(r, 3) + 1 / r)) + dUdmiu * (3 * x * pow(z, 2) / pow(r, 5) - x / pow(r, 3)) + 0 + 0;
    *J4 = d2Udr2 * (pow(y, 2) / pow(r, 2)) + dUdr * (-pow(y, 2) / pow(r, 3) + 1 / r) + d2Udmiu2 * (-y * z / pow(r, 3) * (-y * z / pow(r, 3))) + dUdmiu * (3 * pow(y, 2) * z / pow(r, 5) - z / pow(r, 3)) + d2Udlambda2 * (pow(x / (pow(x, 2) + pow(y, 2)), 2)) + dUdlambda * (-2 * y * x / pow(pow(x, 2) + pow(y, 2), 2));
    *J5 = d2Udr2 * (y * z / pow(r, 2)) + dUdr * (-y * z / pow(r, 3)) + d2Udmiu2 * (-y * z / pow(r, 3) * (-pow(z, 2) / pow(r, 3) + 1 / r)) + dUdmiu * (3 * y * pow(z, 2) / pow(r, 5) - y / pow(r, 3)) + 0 + 0;
    *J6 = d2Udr2 * (pow(z, 2) / pow(r, 2)) + dUdr * (-pow(z, 2) / pow(r, 3) + 1 / r) + d2Udmiu2 * (pow(-pow(z, 2) / pow(r, 3) + 1 / r, 2)) + dUdmiu * (3 * pow(z, 3) / pow(r, 5) - 3 * z / pow(r, 3)) + 0 + 0;
  
}

__global__ void dCreatetm(double *d_tm, int m) {
    int i = threadIdx.x;
	d_tm[i]=-cos(double(i)/(m-1)*pi);
}

__global__ void dCreatebasfunq(double*d_q, double* d_tm) {

    int i = threadIdx.x;
    int j = threadIdx.y;
    int jnum = blockDim.y;
    d_q[i * jnum + j] = cos(j * acos(d_tm[i]));
    
}

__global__ void dCreatebasfunqex(double* d_q, double* d_tm) {
 
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y,inum=blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * (inum * bjnum * jnum) + bj * jnum + i *bjnum* jnum + j;
    int indexi = bi * inum + i;
    int indexj = bj * jnum + j;
    d_q[index] = cos(indexj * acos(d_tm[indexi]));

}

__global__ void dCreatebasfunfqv0(double* d_fq, double* d_tm,int m) {//���������ȴ洢��fq
    int i = threadIdx.x;
    int j = threadIdx.y;
    int jnum = blockDim.y;
    if (j == 0) {
        d_fq[i * jnum + j] = (d_tm[i] + 1)/2;
    }
    else if (j==1) {
        d_fq[i * jnum + j] = (pow(d_tm[i], 2) - 1) / 2;
    }
    else if (j == m-1) {
        d_fq[i * jnum + j] = (cos((j + 1) * acos(d_tm[i])) - cos((j + 1) * acos(-1.0))) / (4 * j + 4) - (cos((j - 1) * acos(d_tm[i])) - cos((j - 1) * acos(-1.0))) / (4 * j - 4);
    }
    else {
        d_fq[i * jnum + j] = (cos((j+1) * acos(d_tm[i])) - cos((j+1) * acos(-1.0))) / (2 * j+2) - (cos((j - 1) * acos(d_tm[i])) - cos((j - 1) * acos(-1.0))) / (2 * j - 2);
    }
    
}

__global__ void dCreatebasfunfq(double* d_fq, double* d_tm, int m) {//���������ȴ洢��fq
    int i = threadIdx.x;
    int j = threadIdx.y;
    int inum = blockDim.x;
    if (j == 0) {
        d_fq[j * inum + i] = (d_tm[i] + 1) / 2;
    }
    else if (j == 1) {
        d_fq[j * inum + i] = (pow(d_tm[i], 2) - 1) / 2;
    }
    else if (j == m - 1) {
        d_fq[j * inum + i] = (cos((j + 1) * acos(d_tm[i])) - cos((j + 1) * acos(-1.0))) / (4 * j + 4) - (cos((j - 1) * acos(d_tm[i])) - cos((j - 1) * acos(-1.0))) / (4 * j - 4);
    }
    else {
        d_fq[j * inum + i] = (cos((j + 1) * acos(d_tm[i])) - cos((j + 1) * acos(-1.0))) / (2 * j + 2) - (cos((j - 1) * acos(d_tm[i])) - cos((j - 1) * acos(-1.0))) / (2 * j - 2);
    }

}

__global__ void dCreatebasfunfqexv0(double* d_fq, double* d_tm, int m) {
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * (inum * bjnum * jnum) + bj * jnum + i * bjnum * jnum + j;
    int indexi = bi * inum + i;
    int indexj = bj * jnum + j;
    if (indexj == 0) {
        d_fq[index] = (d_tm[indexi] + 1) / 2;
    }
    else if (indexj == 1) {
        d_fq[index] = (pow(d_tm[indexi], 2) - 1) / 2;
    }
    else if (indexj == m - 1) {
        d_fq[index] = (cos((indexj + 1) * acos(d_tm[indexi])) - cos((indexj + 1) * acos(-1.0))) / (4 * indexj + 4) - (cos((indexj - 1) * acos(d_tm[indexi])) - cos((indexj - 1) * acos(-1.0))) / (4 * indexj - 4);
    }
    else {
        d_fq[index] = (cos((indexj + 1) * acos(d_tm[indexi])) - cos((indexj + 1) * acos(-1.0))) / (2 * indexj + 2) - (cos((indexj - 1) * acos(d_tm[indexi])) - cos((indexj - 1) * acos(-1.0))) / (2 * indexj - 2);
    }
}

__global__ void dCreatebasfunfqex(double* d_fq, double* d_tm, int m) {
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int indexi = bi * inum + i;
    int indexj = bj * jnum + j;
    int index = indexi + indexj * binum * inum;
    if (indexj == 0) {
        d_fq[index] = (d_tm[indexi] + 1) / 2;
    }
    else if (indexj == 1) {
        d_fq[index] = (pow(d_tm[indexi], 2) - 1) / 2;
    }
    else if (indexj == m - 1) {
        d_fq[index] = (cos((indexj + 1) * acos(d_tm[indexi])) - cos((indexj + 1) * acos(-1.0))) / (4 * indexj + 4) - (cos((indexj - 1) * acos(d_tm[indexi])) - cos((indexj - 1) * acos(-1.0))) / (4 * indexj - 4);
    }
    else {
        d_fq[index] = (cos((indexj + 1) * acos(d_tm[indexi])) - cos((indexj + 1) * acos(-1.0))) / (2 * indexj + 2) - (cos((indexj - 1) * acos(d_tm[indexi])) - cos((indexj - 1) * acos(-1.0))) / (2 * indexj - 2);
    }

}

__global__ void dCreatebasfundqv0(double* d_dq, double* d_tm, int m) {//���������ȴ洢��dq
    int i = threadIdx.x;
    int j = threadIdx.y;
    int jnum = blockDim.y;
    double a = (j%(m-1)==0) ? 0.5 : 1;
    if (i == 0) {
        d_dq[i * jnum + j] = pow(-1.0,j+1)*j*j*a;
    }
    else if (i == m-1) {
        d_dq[i * jnum + j] = j*j*a;
    }
    else {
        d_dq[i * jnum + j] = (j * sin(j * acos(d_tm[i]))) / pow((1 - pow(d_tm[i],2)),0.5)*a;
    }
}

__global__ void dCreatebasfundq(double* d_dq, double* d_tm, int m) {//���������ȴ洢��dq
    int i = threadIdx.x;
    int j = threadIdx.y;
    int inum = blockDim.x;
    double a = (j % (m - 1) == 0) ? 0.5 : 1;
    if (i == 0) {
        d_dq[j * inum + i] = pow(-1.0, j + 1) * j * j * a;
    }
    else if (i == m - 1) {
        d_dq[j * inum + i] = j * j * a;
    }
    else {
        d_dq[j * inum + i] = (j * sin(j * acos(d_tm[i]))) / pow((1 - pow(d_tm[i], 2)), 0.5) * a;
    }
}

__global__ void dCreatebasfundqexv0(double* d_dq, double* d_tm, int m) {
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * (inum * bjnum * jnum) + bj * jnum + i * bjnum * jnum + j;
    int indexi = bi * inum + i;
    int indexj = bj * jnum + j;
    double a = (indexj % (m - 1) == 0) ? 0.5 : 1;
    if (indexi == 0) {
        d_dq[index] = pow(-1.0, indexj + 1) * indexj * indexj * a;
    }
    else if (indexi == m - 1) {
        d_dq[index] = indexj * indexj * a;
    }
    else {
        d_dq[index] = (indexj * sin(indexj * acos(d_tm[indexi]))) / pow((1 - pow(d_tm[indexi], 2)), 0.5) * a;
    }
}

__global__ void dCreatebasfundqex(double* d_dq, double* d_tm, int m) {
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int indexi = bi * inum + i;
    int indexj = bj * jnum + j;
    int index = indexi + indexj * binum * inum;
    double a = (indexj % (m - 1) == 0) ? 0.5 : 1;
    if (indexi == 0) {
        d_dq[index] = pow(-1.0, indexj + 1) * indexj * indexj * a;
    }
    else if (indexi == m - 1) {
        d_dq[index] = indexj * indexj * a;
    }
    else {
        d_dq[index] = (indexj * sin(indexj * acos(d_tm[indexi]))) / pow((1 - pow(d_tm[indexi], 2)), 0.5) * a;
    }
}

__global__ void dCreateqnr(double* d_qnr, int m) {
    int i = threadIdx.x;
    if (i == 0) {
        d_qnr[i] = 1.0/(m-1);
    }
    else if (i == m - 1) {
        d_qnr[i] = 1.0 / (m - 1);
    }
    else {
        d_qnr[i] = 2.0 / (m - 1);
    }

}

__global__ void dCreatet(double *d_t,double *d_tm,double ti,double dt) {
    int i = threadIdx.x;
	d_t[i] = (d_tm[i]*dt+2*ti+dt)/2;
}

__global__ void dCreateGx(double *d_Gx,double *d_y_oldv,double dt,int m) {
    int i = threadIdx.x;
    d_Gx[i] = d_y_oldv[i]*dt/2;
    d_Gx[m+i] = d_y_oldv[m+i] * dt/2;
    d_Gx[2*m + i] = d_y_oldv[2*m + i] * dt/2;
}

__global__ void dCreateGvA(double* d_Gv, double* d_A, double* d_y_oldx, double dt ,int m,double u, double R, double* d_C, double* d_S, double* d_fac) {
    int i = threadIdx.x;
    double f1 = 0, f2 = 0, f3 = 0;
    double A1 = 0, A2 = 0, A3 = 0, A4 = 0, A5 = 0, A6 = 0;
    full_model(d_y_oldx[i], d_y_oldx[m + i], d_y_oldx[2 * m + i], u, R, d_C, d_S, d_fac, &f1, &f2, &f3, &A1, &A2, &A3, &A4, &A5, &A6);
    d_Gv[i] = f1 * dt / 2;
    d_Gv[m + i] = f2 * dt / 2;
    d_Gv[2 * m + i] = f3 * dt / 2;
    d_A[i] = A1 * dt / 2;
    d_A[m + i] = A2 * dt / 2;
    d_A[2 * m + i] = A3 * dt / 2;
    d_A[3 * m + i] = A4 * dt / 2;
    d_A[4 * m + i] = A5 * dt / 2;
    d_A[5 * m + i] = A6 * dt / 2;
}

__global__ void dCreateA(double* d_A, double* d_y_old,  int m, double dt, double u) {
    int i = threadIdx.x;
    double r = pow((pow(d_y_old[i], 2) + pow(d_y_old[m + i], 2) + pow(d_y_old[2 * m + i], 2)), 2.5)*2/dt;
    d_A[i] = -1 * u * (pow(d_y_old[m + i], 2) + pow(d_y_old[2 * m + i], 2) - 2 * pow(d_y_old[i], 2)) / r;
    d_A[m + i] = 3 * u *d_y_old[i]  * d_y_old[m + i]/ r;
    d_A[2 * m + i] = 3 * u * d_y_old[i] * d_y_old[2*m + i] / r;
    d_A[3 * m + i] = -1 * u * (pow(d_y_old[i], 2) + pow(d_y_old[2 * m + i], 2) - 2 * pow(d_y_old[m+i], 2)) / r;
    d_A[4 * m + i] = 3 * u * d_y_old[m+i] * d_y_old[2 * m + i] / r;
    d_A[5 * m + i] = -1 * u * (pow(d_y_old[i], 2) + pow(d_y_old[m + i], 2) - 2 * pow(d_y_old[2*m + i], 2)) / r;
}

__global__ void dCreateJ(double* d_J, double* d_AA, double* d_Te, double* d_P, int m, double dt) {
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * inum + bj * binum * inum * jnum + i + j * binum * inum;
    int pa = (bi == bj) ? -1 : 0;
    double ta = ((bj - 3) == bi) ? dt / 2 : 0;
    int Aa = (bi > 2) ? 1 : 0;
    int Ab = (bj < 3) ? 1 : 0;
    int Ac = (bi - 3) + bj + ((bj - 3) && bj);
    int Ad = (Ac > 0 && Ac < 6) ? Ac : 0;
    d_J[index] = pa * d_P[i + j * inum] + ta * d_Te[i + j * inum] + Aa * Ab * d_AA[Ad * m * m + i + j * inum];
}

__global__ void dCreateJC(double* d_J, double* d_Te, double* d_P, int m, double dt) {
    int mul = m / 32;
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * inum + bj * binum * inum * jnum + i + j * binum * inum;
    int myi = i + inum * (bi % mul);
    int myj = j + jnum * (bj % mul);
    int mybi = bi / mul;
    int mybj = bj / mul;
    int pa = (mybi == mybj) ? -1 : 0;
    double ta = ((mybj - 3) == mybi) ? dt / 2 : 0;
    d_J[index] = pa * d_P[myi + myj * inum * mul] + ta * d_Te[myi + myj * inum * mul];
}

__global__ void dCreateJex(double* d_J, double* d_AA, double* d_Te, double* d_P, int m, double dt) {//����
    int mul = m / 32;
    int i = threadIdx.x; int bi = blockIdx.x;
    int j = threadIdx.y; int bj = blockIdx.y;
    int jnum = blockDim.y, inum = blockDim.x; int bjnum = gridDim.y, binum = gridDim.x;
    int index = bi * inum + bj * binum * inum * jnum + i + j * binum * inum;
    int myi = i + inum * (bi % mul);
    int myj = j + jnum * (bj % mul);
    int mybi = bi / mul;
    int mybj = bj / mul;
    int pa = (mybi == mybj) ? -1 : 0;
    double ta = ((mybj - 3) == mybi) ? dt / 2 : 0;
    int Aa = (mybi > 2) ? 1 : 0;
    int Ab = (mybj < 3) ? 1 : 0;
    int Ac = (mybi - 3) + mybj + ((mybj - 3) && mybj);
    int Ad = (Ac > 0 && Ac < 6) ? Ac : 0;
    d_J[index] = pa * d_P[myi + myj * inum*mul] + ta * d_Te[myi + myj * inum*mul] + Aa * Ab * d_AA[Ad * m * m + myi + myj * inum*mul];
}

__global__ void dCreatey0(double* d_y0, double x0,double y0, double z0, double vx0,double vy0,double vz0,int m) {
    int i = threadIdx.x;
    d_y0[i] = x0;
    d_y0[m+i] = y0;
    d_y0[2*m+i] = z0;
    d_y0[3*m+i] = vx0;
    d_y0[4*m+i] = vy0;
    d_y0[5*m+i] = vz0;
}


/////////////////////////////////// ����Ĵ�����CUDA�˺�����������Ǳ�¶�������Ľӿ�/////////////////////////////////////



extern void hCreatetm(double* d_tm, int m)
{
    hipError_t cudaStatus;
    dCreatetm << <1, m >> > (d_tm, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatetm launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfunq(double* d_q, double* d_tm, int m)
{
    hipError_t cudaStatus;
    dim3 block(m, m);
    dCreatebasfunq << <1, block >> > (d_q, d_tm);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfunq launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfunqex(double* d_q, double* d_tm, int m)
{
    hipError_t cudaStatus;
    int n = (m < 32) ? m : 32;
    dim3 grid(m/32,m/32),block(n, n);
    dCreatebasfunqex << <grid, block >> > (d_q, d_tm);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfunq launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfunfq(double* d_fq, double* d_tm, int m)
{
    hipError_t cudaStatus;
    dim3 block(m, m);
    dCreatebasfunfq << <1, block >> > (d_fq, d_tm, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfunfq launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfunfqex(double* d_fq, double* d_tm, int m)
{
    hipError_t cudaStatus;
    int n = (m < 32) ? m : 32;
    dim3 grid(m/32, m/32), block(n, n);
    dCreatebasfunfqex << <grid, block >> > (d_fq, d_tm, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfunfqex launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfundq(double* d_dq, double* d_tm, int m)
{
    hipError_t cudaStatus;
    dim3 block(m, m);
    dCreatebasfundq << <1, block >> > (d_dq, d_tm, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfundq launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatebasfundqex(double* d_dq, double* d_tm, int m)
{
    hipError_t cudaStatus;
    int n = (m < 32) ? m : 32;
    dim3 grid(m/32, m/32), block(n, n);
    dCreatebasfundqex << <grid, block >> > (d_dq, d_tm, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatebasfundqex launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreateqnr(double* d_qnr,int m)
{
    hipError_t cudaStatus;
    dCreateqnr << <1, m >> > (d_qnr, m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateqnr launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatet(double* d_t, double* d_tm, int m, double ti, double dt)
{
    hipError_t cudaStatus;
    dCreatet << <1, m >> > (d_t, d_tm, ti, dt);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatet launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreateGA(double* d_G, double* d_A, double* d_y_old,int m,double dt,double u, double R, double* d_C, double* d_S, double* d_fac)
{
    hipError_t cudaStatus;
    dCreateGx << <1, m >> > (d_G, &d_y_old[3*m],dt,m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateGx launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    dCreateGvA << <1, m >> > (&d_G[3*m], d_A, d_y_old, dt,m,u,R,d_C, d_S, d_fac);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateGvA launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreateA(double* d_A, double* d_y_old, int m, double dt, double u)
{
    hipError_t cudaStatus;
    dCreateA << <1, m >> > (d_A, d_y_old, m,dt,u);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateA launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}
extern void hCreateJ(double* d_J, double* d_AA, double* d_Te, double* d_P, int m,int n, double dt)
{
    hipError_t cudaStatus;
    dim3 grid(n, n), block(m,m);
    dCreateJ << <grid, block >> > (d_J,d_AA,d_Te,d_P,m,dt);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateJ launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreateJC(double* d_J, double* d_Te, double* d_P, int m, int n, double dt)
{
    hipError_t cudaStatus;
    int bm = (m < 32) ? m : 32;
    dim3 grid(n * m / 32, n * m / 32), block(bm, bm);
    dCreateJC << <grid, block >> > (d_J, d_Te, d_P, m, dt);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateJC launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreateJex(double* d_J, double* d_AA, double* d_Te, double* d_P, int m, int n, double dt)
{
    hipError_t cudaStatus;
    int bm = (m < 32) ? m : 32;
    dim3 grid(n*m/32, n*m/32), block(bm, bm);
    dCreateJex << <grid, block >> > (d_J, d_AA, d_Te, d_P, m, dt);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreateJex launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

extern void hCreatey0(double* d_y0,double x0, double y0, double z0, double vx0, double vy0, double vz0, int m)
{
    hipError_t cudaStatus;
    dCreatey0 << <1, m >> > (d_y0, x0,y0,z0,vx0,vy0,vz0,m);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hCreatey0 launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

